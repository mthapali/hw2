#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 256

const double DIM = 2.0 * cutoff;

int num, tot_num;
double dim;

extern double size;
//
//  benchmarking program
//

__device__ int locationToID(double x, double y, double dim, int num) {
  int xID = x / dim;
  int yID = y / dim;
  return xID * num + yID; 
}

__device__ int locationToID(particle_t &particle, double dim, int num) {
  int xID = particle.x / dim;
  int yID = particle.y / dim;
  return xID * num + yID; 
}

__global__ void assign_particles(int n, particle_t * particles, int* d_next, int* d_grids, double dim, int num) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;
  
  int k = locationToID(particles[tid], dim, num);
  d_next[tid] = atomicExch(&d_grids[k], tid);
}

__global__ void clear_grids(int tot_num, int* d_grids) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= tot_num) return;
  
  d_grids[tid] = -1;
}

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__device__ void compute_self_grid_forces(int tid, particle_t * particles, int * d_next, int head) {
  particle_t* p = &particles[tid];
  for(int i = head; i != -1; i = d_next[i]) {
    if(i != tid)
      apply_force_gpu(*p, particles[i]);
  }
}

__device__ void compute_grid_forces(int tid, particle_t * particles, int * d_next, int head) {
  particle_t* p = &particles[tid];
  for(int i = head; i != -1; i = d_next[i]) {
      apply_force_gpu(*p, particles[i]);
  }
}

__global__ void compute_forces_gpu(int n, particle_t * particles, int * d_next, int * d_grids, double dim, int num)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
  int xID = p->x / dim;
  int yID = p->y / dim;
  int k = xID * num + yID;  
  
  p->ax = p->ay = 0;
  
  // check self
  compute_self_grid_forces(tid, particles, d_next, d_grids[k]);
  
  // check other
  if(xID > 0) {
    compute_grid_forces(tid, particles, d_next, d_grids[k - num]);
    if(yID > 0)
      compute_grid_forces(tid, particles, d_next, d_grids[k - num - 1]);
    if(yID < num - 1)
      compute_grid_forces(tid, particles, d_next, d_grids[k - num + 1]);
  }
  if(xID < num - 1) {
    compute_grid_forces(tid, particles, d_next, d_grids[k + num]);
    if(yID > 0)
      compute_grid_forces(tid, particles, d_next, d_grids[k + num - 1]);
    if(yID < num - 1)
      compute_grid_forces(tid, particles, d_next, d_grids[k + num + 1]);
  }
  if(yID > 0) compute_grid_forces(tid, particles, d_next, d_grids[k - 1]);
  if(yID < num - 1) compute_grid_forces(tid, particles, d_next, d_grids[k + 1]);
}

__global__ void compute_grid_forces_gpu(particle_t * particles, int * d_next,int tot_num, int * d_grids, double dim, int num)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= tot_num) return;

  int xID = tid / num;
  int yID = tid % num;
  int k = tid;

  for(int i = d_grids[tid]; i != -1; i = d_next[i]) {
    particle_t * p = &particles[i];

    p->ax = p->ay = 0;
  
    // check self
    compute_self_grid_forces(i, particles, d_next, d_grids[k]);
  
    // check other
    if(xID > 0) {
      compute_grid_forces(i, particles, d_next, d_grids[k - num]);
      if(yID > 0)
        compute_grid_forces(i, particles, d_next, d_grids[k - num - 1]);
      if(yID < num - 1)
        compute_grid_forces(i, particles, d_next, d_grids[k - num + 1]);
    }
    if(xID < num - 1) {
      compute_grid_forces(i, particles, d_next, d_grids[k + num]);
      if(yID > 0)
        compute_grid_forces(i, particles, d_next, d_grids[k + num - 1]);
      if(yID < num - 1)
        compute_grid_forces(i, particles, d_next, d_grids[k + num + 1]);
    }
    if(yID > 0) compute_grid_forces(i, particles, d_next, d_grids[k - 1]);
    if(yID < num - 1) compute_grid_forces(i, particles, d_next, d_grids[k + 1]);
  }
}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}

int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    init_particles( n, particles );

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;
    
    
    // set up grids
    double set_grid_time = read_timer();
    
    num = (int)ceil(size*1.0 / DIM); // we get the num of the grid for one directions
    tot_num = num * num; // total number of grids
    dim = size/num; // the acutal size of a subgrid
    
    int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int g_blks = (tot_num + NUM_THREADS - 1) / NUM_THREADS;
    
    
    int * d_grids;
    hipMalloc((void **) &d_grids, tot_num * sizeof(int));
    int * d_next;
    hipMalloc((void **) &d_next, n * sizeof(int));
    
    hipDeviceSynchronize();
    
  clear_grids <<< g_blks, NUM_THREADS >>> (tot_num, d_grids);
    
  assign_particles <<< blks, NUM_THREADS >>> (n, d_particles, d_next, d_grids, dim, num);
  
    hipDeviceSynchronize();
    
    set_grid_time = read_timer() - set_grid_time;
    
    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //

  
  compute_forces_gpu <<< blks, NUM_THREADS >>> (n, d_particles, d_next, d_grids, dim, num);
  //compute_grid_forces_gpu <<< g_blks, NUM_THREADS >>> (d_particles, d_next, tot_num, d_grids, dim, num);
        
        //
        //  move particles
        //
  move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
       
   // Re-locate all the particles to grids
   
   clear_grids <<< g_blks, NUM_THREADS >>> (tot_num, d_grids);
   assign_particles <<< blks, NUM_THREADS >>> (n, d_particles, d_next, d_grids, dim, num);    
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
      // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
  }
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    hipFree(d_grids);
    hipFree(d_next);
    if( fsave )
        fclose( fsave );
    
    return 0;
}